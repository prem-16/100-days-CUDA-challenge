#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vecADDKernel(float *A, float *B, float *C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
        printf("No CUDA-compatible GPU found!\n");
    }
    else
    {
        printf("CUDA device count: %d\n", deviceCount);
    }
    const int N = 2 << 20;              // elements in vec
    const int size = N * sizeof(float); // total size of vectors in bytes

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // initialize with values in host
    for (int i = 0; i < N; i++)
    {
        h_A[i] = 1;
        h_B[i] = 1;
        h_C[i] = 0;
    }

    float *d_A, *d_B, *d_C;
    // Allocate device memory for d_A and d_B
    hipMalloc((void **)&d_A, size);
    // copy h_A host to d_A device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_C, size);

    // invoke kernel code
    vecADDKernel<<<ceil(N / 256.0), 256>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // free device memory for A,B, C
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceSynchronize();
    // print first 10 element
    for (int i = 0; i < 10; i++)
    {
        printf("array C[%d] =%f\n", i, h_C[i]);
    }
    return 0;
}